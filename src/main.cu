#include "hip/hip_runtime.h"
#include <iostream>

#include "Colour.hpp"
#include "Vec3.hpp"
#include "Ray.hpp"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 rayColour(const Ray& r) {
   vec3 unit_direction = unit_vector(r.direction());
   float t = 0.5f*(unit_direction.y() + 1.0f);
   return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
}

__global__ void render(vec3 *fb, int max_x, int max_y, vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin) {
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;
   if((i >= max_x) || (j >= max_y)) return;
   int pixel_index = j*max_x + i;
   float u = float(i) / float(max_x);
   float v = float(j) / float(max_y);
   Ray r(origin, lower_left_corner + u*horizontal + v*vertical);
   fb[pixel_index] = rayColour(r);
}

int main(int argc, char** argv) {
    //Image
    double aspect_ratio = 16.0 / 9.0;
    int const imageWidth = 1024;
    int const imageHeight = static_cast<int>(imageWidth / aspect_ratio);
    int numPixels = imageHeight * imageWidth;

    // Allocate frame buffer (fb)
    size_t fb_size = numPixels*sizeof(vec3);
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // Build frame buffer
    int tx = 8;
    int ty = 8;

    //render buf
    dim3 blocks(imageWidth/tx + 1, imageHeight/ty + 1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(
        fb, imageWidth, imageHeight,
        vec3(-2.0, -1.0, -1.0),
        vec3(4.0, 0.0, 0.0),
        vec3(0.0, 2.0, 0.0),
        vec3(0.0, 0.0, 0.0));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render
    std::cout << "P3\n"
        << imageWidth
        << " "
        << imageHeight
        << "\n255"
        << std::endl;

    for (int j = imageHeight-1; j >= 0; j--) {
        std::cerr << "Scanlines remaining: "
            << j
            << " "
            << std::endl
            << std::flush;

        for (int i = 0; i < imageWidth; i++) {
            size_t pixel_idx = j*imageWidth + i;

            write_colour(std::cout, fb[pixel_idx], 1);
        }
    }

    std::cerr << "\nDone!" << std::endl;

    return 0;
}